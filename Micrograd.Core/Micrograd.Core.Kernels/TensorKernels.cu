
#include <hip/hip_runtime.h>
extern "C" __global__ void addVect(const float* a, const float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) 
        c[i] = a[i] + b[i];       
}

extern "C" __global__ void MulVect(const float* a, const float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) 
        c[i] = a[i] * b[i];
}

extern "C" __global__ void TanhVect(const float* input, float* output, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) 
        output[i] = tanhf(input[i]);
}

extern "C" __global__ void ReLUVect(const float* input, float* output, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) 
        output[i] = fmaxf(0.0f, input[i]);
}

extern "C" __global__ void MatMulVect(const float* A, const float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

extern "C" __global__ void UpdateVect(float* params, const float* gradients, float learningRate, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) 
        params[i] -= learningRate * gradients[i];
}